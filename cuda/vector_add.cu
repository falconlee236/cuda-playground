#include <cstdlib>
#include <random>
#include <iostream>

#include "hip/hip_runtime.h"

#define N 512

std::random_device rd; // hardware based seed
std::mt19937 gen(rd()); // mersenne twister engine
std::uniform_int_distribution<> dis(0, 99); // 0~99 distribution

void random_ints(int* des, int size){
    for(int i = 0; i < size; i++){
        des[i] = dis(gen);
    }
}

__global__ void add(int *a, int *b, int *c){
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main(void){
    int *a, *b, *c; // host
    int *d_a, *d_b, *d_c; // device
    int size = N * sizeof(int);

    // allocate device(gpu) memory
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_b, size);

    // allocate host memory
    a = (int *)malloc(size);  random_ints(a, N);
    b = (int *)malloc(size);  random_ints(b, N);
    c = (int *)malloc(size);

    // host values
    std::cout << "a:\n";
    for(int i = 0; i < size; i++){
        std::cout << a[i] << " ";
    }
    std::cout << "b:\n";
    for(int i = 0; i < size; i++){
        std::cout << b[i] << " ";
    }

    // copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // launch add() kernel on gpu with n blocks
    add<<<N,1>>>(d_a, d_b, d_c);

    // copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c); 
    return 0;
}